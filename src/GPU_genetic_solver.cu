#include "hip/hip_runtime.h"
#include "GPU_genetic_solver.hpp"
#include <algorithm>
#include <random>
#include <set>
#include <queue>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, const int line) {
	if (err != hipSuccess) {
        std::string errMsg = 
			string(hipGetErrorString(err)) +
			" (file:" + string(file) +
			" at line:" + to_string(line) + ")";
        throw std::runtime_error(errMsg);
    }
}

GPU_genetic_solver::GPU_genetic_solver(int2 *dev_map, int map_size, int *dev_population, int population_size, int number_path_crossover, int number_path_mutation) :
	dev_map(dev_map),
	map_size(map_size),
	dev_population(dev_population),
	population_size(population_size),
	number_path_crossover(number_path_crossover),
	number_path_mutation(number_path_mutation) {

}

GPU_genetic_solver * GPU_genetic_solver::create(const Map & map, int population_size, float rate_path_crossover, float rate_path_mutation) {

	
	/* TODO on GPU */
	vector<Path> population;
	for (int i = 0; i < population_size; i++) {
		population.push_back(Path::random(map));
	}
	sort(population.begin(), population.end(), [&map](const Path & p1, const Path & p2) {
		return fitness(map, p1) < fitness(map, p2);
	});
	/*          */
	

	int map_size = map.number_cities();
	int path_length = map_size + 1;
	
	// Create map on GPU
	vector<int2> host_map;
	for (int i = 0; i < map.number_cities(); i++) {
		City city = map[i];
		host_map.push_back(int2 {city.getX(), city.getY()});
	}
	int2 *dev_map = nullptr;
	HANDLE_ERROR(hipMalloc(&dev_map, host_map.size()*sizeof(int2)));
	HANDLE_ERROR(hipMemcpy(dev_map, host_map.data(), host_map.size()*sizeof(int2), hipMemcpyHostToDevice));
	

	
	// Create paths on GPU
	vector<int> host_population;
	for (int i = 0; i < population_size; i++) {
		host_population.insert(host_population.end(), population[i].begin(), population[i].end());
	}
	int *dev_population = nullptr; 
	HANDLE_ERROR(hipMalloc(&dev_population, population_size*path_length*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(dev_population, host_population.data(), host_population.size()*sizeof(int), hipMemcpyHostToDevice));
	

	return new GPU_genetic_solver(dev_map, map_size, dev_population, population_size, population_size*rate_path_crossover, population_size*rate_path_mutation);
}

GPU_genetic_solver::~GPU_genetic_solver() {
	HANDLE_ERROR(hipFree(nullptr));
	HANDLE_ERROR(hipFree(dev_population));
}

int GPU_genetic_solver::fitness(const Map & map, const Path & path) {
	int d = 0;
	for (int i = 0; i < path.number_cities()-1; i++) {
		d += map[path[i]].distance(map[path[i+1]]);
	}
	return d;
}

Path GPU_genetic_solver::cross_over(const Path & path1, const Path & path2) {
	mt19937 generator(random_device{}());
	int n = path1.number_cities();
	
	
	/* */
	// OX crossover
	Path path = Path(vector<int>(n));
	int i = uniform_int_distribution<int>(1, n-3)(generator);
	int j = uniform_int_distribution<int>(i+1, n-2)(generator);
	set<int> seen;
	for (int k = i; k <= j; k++) {
		path[k] = path1[i];
		seen.insert(path1[i]);
	}
	queue<int> not_seen;
	for (int k = 0; k < n; k++) {
		if (seen.find(path2[k]) == seen.end()) {
			not_seen.push(path2[k]);
		}
	}
	for (int k = 0; k < i; k++) {
		path[k] = not_seen.front();
		not_seen.pop();
	}
	for (int k = i+1; k < n; k++) {
		path[k] = not_seen.front();
		not_seen.pop();
	}
	/* */
	
	
	/*
	Path path = Path(vector<int>(n));
	int i = uniform_int_distribution<int>(1, n-2)(generator);
	int first = path1[i];
	int previous = path2[i];
	path[i] = path1[i];
	while (previous != first) {
		if (path1[i] == previous) {
			path[i] = path1[i];
			previous = path2[i];
		}
		i = (i+1)%n;
	}
	for (i = 0 ; i < n ; i++) {
		if (path[i] == 0) {
			path[i] = path2[i];
		}
    }
	*/

	return path;
}

void GPU_genetic_solver::mutation(Path & path) {
	// 2-OPT mutation
	mt19937 generator(random_device{}());
	int n = path.number_cities();
	int i = uniform_int_distribution<int>(1, n-3)(generator);
	int j = uniform_int_distribution<int>(i+1, n-2)(generator);
	while (i < j) {
		swap(path[i], path[j]);
		i++;
		j--;
	}
}

void GPU_genetic_solver::optimize() {
	/* TODO RM */
	/* GPU to HOST */
	int path_length = map_size+1;
	
	
	vector<int> host_path_1(population_size*path_length);
	HANDLE_ERROR(hipMemcpy(host_path_1.data(), dev_population, host_path_1.size()*sizeof(int), hipMemcpyDeviceToHost));
	vector<Path> population;
	for (int i = 0; i < population_size; i++){
		auto begin_iter = host_path_1.begin() + i*path_length;
		Path path(vector<int>(begin_iter, begin_iter + path_length));
		population.push_back(path);
	}
	
	/* TODO RM */
	/* GPU to HOST */
	vector<int2> host_map(map_size);
	HANDLE_ERROR(hipMemcpy(host_map.data(), dev_map, map_size*sizeof(int2), hipMemcpyDeviceToHost));
	vector<City> cities;
	for (int2 point : host_map) {
		cities.push_back(City(point.x, point.y));
	}
	Map map(cities);
	
	
	
	vector<Path> childs_population;
	for (unsigned int i = 0; i <  population.size(); i++) {
		childs_population.push_back(cross_over(population[(2*i) % number_path_crossover], population[(2*i + 1) % number_path_crossover]));
	}
	for_each(childs_population.begin(), childs_population.begin()+number_path_mutation, [](Path & p) {
		mutation(p);
	});
	sort(childs_population.begin(), childs_population.end(), [&map](const Path & p1, const Path & p2) {
		return fitness(map, p1) < fitness(map, p2);
	});
	vector<Path> next_population;
	int parent_index = 0;
	int child_index = 0;
	for (unsigned int i = 0; i <  population.size(); i++) {
		if (fitness(map, population[parent_index]) < fitness(map, childs_population[child_index])) {
			next_population.push_back(population[parent_index++]);
		} else {
			next_population.push_back(childs_population[child_index++]);
		}
	}
	population = next_population;


	/* TODO RM */
	/* HOST to GPU */
	vector<int> host_population_2;
	for (int i = 0; i < population.size(); i++) {
		host_population_2.insert(host_population_2.end(), population[i].begin(), population[i].end());
	}
	HANDLE_ERROR(hipMemcpy(dev_population, host_population_2.data(), host_population_2.size()*sizeof(int), hipMemcpyHostToDevice));
}

Path GPU_genetic_solver::get_solution() const {
	int path_length = map_size+1;
	
	
	vector<int> host_path(path_length);
	HANDLE_ERROR(hipMemcpy(host_path.data(), dev_population, path_length*sizeof(int), hipMemcpyDeviceToHost));
	return Path(host_path);
}
