#include "hip/hip_runtime.h"
#include "GPU_genetic_solver.hpp"


#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include <algorithm>
#include <random>
#include <set>
#include <queue>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, const int line) {
	if (err != hipSuccess) {
        std::string errMsg = 
			string(hipGetErrorString(err)) +
			" (file:" + string(file) +
			" at line:" + to_string(line) + ")";
        throw std::runtime_error(errMsg);
    }
}

GPU_genetic_solver::GPU_genetic_solver(int2 *dev_map, int map_size, int *dev_population, int *dev_childs_population, int population_size, int number_path_crossover, int number_path_mutation) :
	dev_map(dev_map),
	map_size(map_size),
	dev_population(dev_population),
	dev_childs_population(dev_childs_population),
	population_size(population_size),
	number_path_crossover(number_path_crossover),
	number_path_mutation(number_path_mutation) {

}

GPU_genetic_solver * GPU_genetic_solver::create(const Map & map, int population_size, float rate_path_crossover, float rate_path_mutation) {

	
	/* TODO on GPU */
	vector<Path> population;
	for (int i = 0; i < population_size; i++) {
		population.push_back(Path::random(map));
	}
	sort(population.begin(), population.end(), [&map](const Path & p1, const Path & p2) {
		return fitness(map, p1) < fitness(map, p2);
	});
	/*          */
	

	int map_size = map.number_cities();
	int path_length = map_size + 1;
	
	// Create map on GPU
	vector<int2> host_map;
	for (int i = 0; i < map.number_cities(); i++) {
		City city = map[i];
		host_map.push_back(int2 {city.getX(), city.getY()});
	}
	int2 *dev_map = nullptr;
	HANDLE_ERROR(hipMalloc(&dev_map, host_map.size()*sizeof(int2)));
	HANDLE_ERROR(hipMemcpy(dev_map, host_map.data(), host_map.size()*sizeof(int2), hipMemcpyHostToDevice));
	

	
	// Create paths on GPU
	vector<int> host_population;
	for (int i = 0; i < population_size; i++) {
		host_population.insert(host_population.end(), population[i].begin(), population[i].end());
	}
	int *dev_population = nullptr; 
	HANDLE_ERROR(hipMalloc(&dev_population, population_size*path_length*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(dev_population, host_population.data(), host_population.size()*sizeof(int), hipMemcpyHostToDevice));
	int *dev_childs_population = nullptr; 
	HANDLE_ERROR(hipMalloc(&dev_childs_population, population_size*path_length*sizeof(int)));



	return new GPU_genetic_solver(dev_map, map_size, dev_population, dev_childs_population, population_size, population_size*rate_path_crossover, population_size*rate_path_mutation);
}

GPU_genetic_solver::~GPU_genetic_solver() {
	HANDLE_ERROR(hipFree(nullptr));
	HANDLE_ERROR(hipFree(dev_population));
}

int GPU_genetic_solver::fitness(const Map & map, const Path & path) {
	int d = 0;
	for (int i = 0; i < path.number_cities()-1; i++) {
		d += map[path[i]].distance(map[path[i+1]]);
	}
	return d;
}

__global__
void cross_over(int *dev_population, int *dev_childs_population, int population_size, int path_length) {

	//TODO
	
	for (int i = 0; i < population_size*path_length; i++) {
		dev_childs_population[i] = dev_population[i];
	}
	
}	

Path GPU_genetic_solver::cross_over_tmp(const Path & path1, const Path & path2) {
	mt19937 generator(random_device{}());
	int n = path1.number_cities();
	
	
	/* */
	// OX crossover
	Path path = Path(vector<int>(n));
	int i = uniform_int_distribution<int>(1, n-3)(generator);
	int j = uniform_int_distribution<int>(i+1, n-2)(generator);
	set<int> seen;
	for (int k = i; k <= j; k++) {
		path[k] = path1[i];
		seen.insert(path1[i]);
	}
	queue<int> not_seen;
	for (int k = 0; k < n; k++) {
		if (seen.find(path2[k]) == seen.end()) {
			not_seen.push(path2[k]);
		}
	}
	for (int k = 0; k < i; k++) {
		path[k] = not_seen.front();
		not_seen.pop();
	}
	for (int k = i+1; k < n; k++) {
		path[k] = not_seen.front();
		not_seen.pop();
	}
	/* */
	
	
	/*
	Path path = Path(vector<int>(n));
	int i = uniform_int_distribution<int>(1, n-2)(generator);
	int first = path1[i];
	int previous = path2[i];
	path[i] = path1[i];
	while (previous != first) {
		if (path1[i] == previous) {
			path[i] = path1[i];
			previous = path2[i];
		}
		i = (i+1)%n;
	}
	for (i = 0 ; i < n ; i++) {
		if (path[i] == 0) {
			path[i] = path2[i];
		}
    }
	*/

	return path;
}

__global__
void mutation(int *dev_population, int population_size, int path_length) {
	
	
	for (int path = 0; path < population_size; path++) {
		
		
		int begin_index = (((dev_population[path]*path) << 2) | 784) % (path_length-3) + 1; // TODO random
		int end_index = (((dev_population[path]*path) << 3) | 993) % (path_length-begin_index-1) + begin_index; // TODO random
		
		int i = path*path_length + begin_index;
		int j = path*path_length + end_index;
		while (i < j) {
			//
			int tmp = dev_population[i];
			dev_population[i] = dev_population[j];
			dev_population[j] = tmp;
			//
			i++;
			j--;
		}
	}
	
}	

void GPU_genetic_solver::mutation_tmp(Path & path) {
	// 2-OPT mutation
	mt19937 generator(random_device{}());
	int n = path.number_cities();
	int i = uniform_int_distribution<int>(1, n-3)(generator);
	int j = uniform_int_distribution<int>(i+1, n-2)(generator);
	while (i < j) {
		swap(path[i], path[j]);
		i++;
		j--;
	}
}

void GPU_genetic_solver::optimize() {
	/* TODO RM */
	/* GPU to HOST */
	int path_length = map_size+1;
	
	
	vector<int> host_path_1(population_size*path_length);
	HANDLE_ERROR(hipMemcpy(host_path_1.data(), dev_population, host_path_1.size()*sizeof(int), hipMemcpyDeviceToHost));
	vector<Path> population;
	for (int i = 0; i < population_size; i++){
		auto begin_iter = host_path_1.begin() + i*path_length;
		Path path(vector<int>(begin_iter, begin_iter + path_length));
		population.push_back(path);
	}
	
	/* TODO RM */
	/* GPU to HOST */
	vector<int2> host_map(map_size);
	HANDLE_ERROR(hipMemcpy(host_map.data(), dev_map, map_size*sizeof(int2), hipMemcpyDeviceToHost));
	vector<City> cities;
	for (int2 point : host_map) {
		cities.push_back(City(point.x, point.y));
	}
	Map map(cities);
	
	
	
	
	cross_over<<<1,1>>>(dev_population, dev_childs_population, population_size, path_length);
	mutation<<<1,1>>>(dev_childs_population, number_path_mutation, path_length);
	
	
	vector<Path> childs_population;
	
	
	// CPU child
	/*
	for (unsigned int i = 0; i <  population.size(); i++) {
		childs_population.push_back(cross_over_tmp(population[(2*i) % number_path_crossover], population[(2*i + 1) % number_path_crossover]));
	}
	for_each(childs_population.begin(), childs_population.begin()+number_path_mutation, [](Path & p) {
		mutation_tmp(p);
	});
	 */

	//GPU child
	/* */
	vector<int> childs_population_host(population_size*path_length);
	HANDLE_ERROR(hipMemcpy(childs_population_host.data(), dev_childs_population, population_size*path_length*sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i < population_size; i++){
		auto begin_iter = childs_population_host.begin() + i*path_length;
		Path path(vector<int>(begin_iter, begin_iter + path_length));
		childs_population.push_back(path);
	}
	/* */
	
	
	sort(childs_population.begin(), childs_population.end(), [&map](const Path & p1, const Path & p2) {
		return fitness(map, p1) < fitness(map, p2);
	});
	vector<Path> next_population;
	int parent_index = 0;
	int child_index = 0;
	for (unsigned int i = 0; i <  population.size(); i++) {
		if (fitness(map, population[parent_index]) < fitness(map, childs_population[child_index])) {
			next_population.push_back(population[parent_index++]);
		} else {
			next_population.push_back(childs_population[child_index++]);
		}
	}
	population = next_population;


	/* TODO RM */
	/* HOST to GPU */
	vector<int> host_population_2;
	for (int i = 0; i < population.size(); i++) {
		host_population_2.insert(host_population_2.end(), population[i].begin(), population[i].end());
	}
	HANDLE_ERROR(hipMemcpy(dev_population, host_population_2.data(), host_population_2.size()*sizeof(int), hipMemcpyHostToDevice));
}

Path GPU_genetic_solver::get_solution() const {
	int path_length = map_size+1;
	
	
	vector<int> host_path(path_length);
	HANDLE_ERROR(hipMemcpy(host_path.data(), dev_population, path_length*sizeof(int), hipMemcpyDeviceToHost));
	return Path(host_path);
}
