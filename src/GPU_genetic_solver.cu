#include "hip/hip_runtime.h"
#include "GPU_genetic_solver.hpp"
#include <algorithm>
#include <random>
#include <set>
#include <queue>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, const int line) {
	if (err != hipSuccess) {
        std::string errMsg = 
			string(hipGetErrorString(err)) +
			" (file:" + string(file) +
			" at line:" + to_string(line) + ")";
        throw std::runtime_error(errMsg);
    }
}

GPU_genetic_solver::GPU_genetic_solver(const Map & map, vector<Path> population, int number_path_crossover, int number_path_mutation, int2 *dev_map, int *dev_population) :
	map(map),
	population(population),
	number_path_crossover(number_path_crossover),
	number_path_mutation(number_path_mutation),
	
	
	dev_map(dev_map),
	dev_population(dev_population) {

}

GPU_genetic_solver * GPU_genetic_solver::create(const Map & map, int population_size, float rate_path_crossover, float rate_path_mutation) {
	int nnn = 10;
	int2 *dev_map = nullptr;
	int *dev_population = nullptr; 
	HANDLE_ERROR(hipMalloc(&dev_map, map.number_cities() * sizeof(int2)));
	HANDLE_ERROR(hipMalloc(&dev_population, nnn * population_size * sizeof(int)));
	//dev_map[0].x;



	vector<Path> population;
	for (int i = 0; i < population_size; i++) {
		population.push_back(Path::random(map));
	}
	sort(population.begin(), population.end(), [&map](const Path & p1, const Path & p2) {
		return fitness(map, p1) < fitness(map, p2);
	});
	return new GPU_genetic_solver(map, population, population_size*rate_path_crossover, population_size*rate_path_mutation, dev_map, dev_population);
}

GPU_genetic_solver::~GPU_genetic_solver() {
	HANDLE_ERROR(hipFree(nullptr));
	HANDLE_ERROR(hipFree(dev_population));
}

int GPU_genetic_solver::fitness(const Map & map, const Path & path) {
	int d = 0;
	for (int i = 0; i < path.number_cities()-1; i++) {
		d += map[path[i]].distance(map[path[i+1]]);
	}
	return d;
}

Path GPU_genetic_solver::cross_over(const Path & path1, const Path & path2) {
	mt19937 generator(random_device{}());
	int n = path1.number_cities();
	
	
	/* */
	// OX crossover
	Path path = Path(vector<int>(n));
	int i = uniform_int_distribution<int>(1, n-3)(generator);
	int j = uniform_int_distribution<int>(i+1, n-2)(generator);
	set<int> seen;
	for (int k = i; k <= j; k++) {
		path[k] = path1[i];
		seen.insert(path1[i]);
	}
	queue<int> not_seen;
	for (int k = 0; k < n; k++) {
		if (seen.find(path2[k]) == seen.end()) {
			not_seen.push(path2[k]);
		}
	}
	for (int k = 0; k < i; k++) {
		path[k] = not_seen.front();
		not_seen.pop();
	}
	for (int k = i+1; k < n; k++) {
		path[k] = not_seen.front();
		not_seen.pop();
	}
	/* */
	
	
	/*
	Path path = Path(vector<int>(n));
	int i = uniform_int_distribution<int>(1, n-2)(generator);
	int first = path1[i];
	int previous = path2[i];
	path[i] = path1[i];
	while (previous != first) {
		if (path1[i] == previous) {
			path[i] = path1[i];
			previous = path2[i];
		}
		i = (i+1)%n;
	}
	for (i = 0 ; i < n ; i++) {
		if (path[i] == 0) {
			path[i] = path2[i];
		}
    }
	*/

	return path;
}

void GPU_genetic_solver::mutation(Path & path) {
	// 2-OPT mutation
	mt19937 generator(random_device{}());
	int n = path.number_cities();
	int i = uniform_int_distribution<int>(1, n-3)(generator);
	int j = uniform_int_distribution<int>(i+1, n-2)(generator);
	while (i < j) {
		swap(path[i], path[j]);
		i++;
		j--;
	}
}

void GPU_genetic_solver::optimize() {
	vector<Path> childs_population;
	for (unsigned int i = 0; i <  population.size(); i++) {
		childs_population.push_back(cross_over(population[(2*i) % number_path_crossover], population[(2*i + 1) % number_path_crossover]));
	}
	for_each(childs_population.begin(), childs_population.begin()+number_path_mutation, [](Path & p) {
		mutation(p);
	});
	sort(childs_population.begin(), childs_population.end(), [this](const Path & p1, const Path & p2) {
		return fitness(this->map, p1) < fitness(this->map, p2);
	});
	vector<Path> next_population;
	int parent_index = 0;
	int child_index = 0;
	for (unsigned int i = 0; i <  population.size(); i++) {
		if (fitness(map, population[parent_index]) < fitness(map, childs_population[child_index])) {
			next_population.push_back(population[parent_index++]);
		} else {
			next_population.push_back(childs_population[child_index++]);
		}
	}
	population = next_population;
}

Path GPU_genetic_solver::get_solution() const {
	return population[0];
}
