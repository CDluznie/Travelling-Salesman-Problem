#include <iostream>
#include "map.hpp"
#include "path.hpp"
#include "solver.hpp"
#include "GPU_genetic_solver.hpp"
#include "drawer.hpp"
#include "SDL_drawer.hpp"

int main() {
	
	int number_cities = 150;
	int x_min = 100, x_max = 999;
	int y_min = 100, y_max = 999;
	
	Map map = Map::random(number_cities, x_min, x_max, y_min, y_max);
	
	int population_size = 128;
	float rate_path_crossover = 0.625;
	float rate_path_mutation = 0.85;
	GPU_genetic_solver * solver = GPU_genetic_solver::create(map, population_size, rate_path_crossover, rate_path_mutation);
	
	Path solution = solver->get_solution();
	
	Drawer * drawer = Drawer::new_SDL_drawer(900, 650, map, x_min, x_max, y_min, y_max);
	
	while (!drawer->stop()) {
		drawer->clean();
		drawer->draw_path(solution);
		drawer->update();
		solver->optimize();
		solution = solver->get_solution();
	}
	
	delete solver;
	delete drawer;
  
    return 0;
    
}


